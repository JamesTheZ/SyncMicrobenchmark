#include "hip/hip_runtime.h"

#include"measurement.cuh"
#include "wrap_launch_functions.cuh"
#include "../share/util.h"

#include <stdio.h>


//1. measure latencys in cycle (single GPU)
//1.1. measure the latency of instructions in the first SM
//1.2. measure the throughput of instructions in the first SM
int measureIntraSMLatency(latencys* result, 
	launchfunction_rkernel run_func, fbaseKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock, 
	float a, float b, unsigned int tile)
{
	{
		int errorcode=1;
		hipError_t e;

		double * d_out;
		unsigned int totalThreadsPerGPU = blockPerGPU*threadPerBlock;
		hipMalloc((void **)&d_out, sizeof(double) * totalThreadsPerGPU*1);	
		unsigned int warp_count=blockPerGPU*threadPerBlock/32;

	 	unsigned int * h_time_stamp = (unsigned int*)malloc(sizeof(unsigned int)*warp_count*2); 
	 	unsigned int * d_time_stamp;
	 	hipMalloc((void**)&d_time_stamp, sizeof(unsigned int)*warp_count*2);

	 	unsigned int * h_idx = (unsigned int*)malloc(sizeof(unsigned int)*warp_count*2); 
	 	unsigned int * d_idx;
	 	hipMalloc((void**)&d_idx, sizeof(unsigned int)*warp_count*2);

		void*KernelArgs[] = {(void*)&a, 
						(void*)&b,
						(void*)&d_out,
						(void*)&d_time_stamp,
						(void*)&d_idx,
						(void*)&tile};

		cudaCheckError(); 

		timespec tsstart,tsend;
		long time_elapsed_ns_lat ;
		double latency_lat[SIZE];
		double latency_max[SIZE];
		double latency_min[SIZE];
		unsigned int ulatency_max;
		unsigned int ulatency_min;
		for(int i=0; i<SIZE; i++)
		{
			//clock
			clock_gettime(CLOCK_REALTIME, &tsstart);
			//launch
			run_func(kernel_func,blockPerGPU,threadPerBlock,KernelArgs,1,NULL);
			hipDeviceSynchronize();
			clock_gettime(CLOCK_REALTIME, &tsend);
			//execution

			hipMemcpy(h_time_stamp, d_time_stamp, sizeof(unsigned int)*warp_count*2, hipMemcpyDeviceToHost);
			hipMemcpy(h_idx, d_idx, sizeof(unsigned int)*warp_count*2, hipMemcpyDeviceToHost);

	 		time_elapsed_ns_lat = (tsend.tv_nsec-tsstart.tv_nsec);
	 		time_elapsed_ns_lat += 1000000000*(tsend.tv_sec-tsstart.tv_sec);
	 		latency_lat[i]=time_elapsed_ns_lat;

	 		GetLatencyOfSM(ulatency_min,ulatency_max,warp_count,h_time_stamp,h_idx,0);
	 		latency_min[i]=ulatency_min;
	 		latency_max[i]=ulatency_max;
			e=hipGetLastError();                                 
	 		if(e!=hipSuccess) {                                              
	   			fprintf(stderr,"Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); 
				errorcode=-1;
				break;
	 		}
		}
		cudaCheckError();

		getStatistics(result->mean_lat, result->s_lat, latency_lat+1, SIZE-1);
		getStatistics(result->latency_min, result->s_latency_min, latency_min+1, SIZE-1);
		getStatistics(result->latency_max, result->s_latency_max, latency_max+1, SIZE-1);

		hipFree(d_out);
		hipFree(d_time_stamp);
		hipFree(d_idx);
		free(h_time_stamp);
		free(h_idx);

		// e=hipGetLastError();                                 
 	// 	if(e!=hipSuccess) {                                              
  //  			fprintf(stderr,"Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); 
		// 	errorcode=-1;
 	// 	}

 		if(errorcode!=1)
 		{
	 		hipDeviceReset();
			return -1;
 		}

		return 1;
	}
}
//2. measure latencys in ns (involve several SMs) TODO
int measureInterSMLatency(latencys* result, 
	launchfunction_rkernel run_func, fbaseKernel kernel_func, 
	unsigned int gpu_count,
	unsigned int blockPerGPU, unsigned int threadPerBlock)
{
	{
		int errorcode=1;
		hipError_t e;

		hipStream_t *mstream = (hipStream_t*)malloc(sizeof(hipStream_t)*gpu_count);
		void***packedKernelArgs = (void***)malloc(sizeof(void**)*gpu_count); 
		hipLaunchParams *launchParamsList = (hipLaunchParams *)malloc(
      		sizeof(hipLaunchParams)*gpu_count);

		float a=2;
		float b=2;
		double **d_out = (double**)malloc(sizeof(double)*gpu_count);
		unsigned int* nptr=NULL;
		unsigned int tile=32;

		for(int deviceid=0; deviceid<gpu_count;deviceid++)
		{
			hipSetDevice(deviceid);
			packedKernelArgs[deviceid]=(void**)malloc(sizeof(void*)*6);

			hipStreamCreate(&mstream[deviceid]);

			cudaCheckError();
			hipMalloc((void**)&d_out[deviceid], sizeof(double));
			packedKernelArgs[deviceid][0]=(void*)&a;
			packedKernelArgs[deviceid][1]=(void*)&b;
			packedKernelArgs[deviceid][2]=(void*)&d_out[deviceid];
			packedKernelArgs[deviceid][3]=(void*)&nptr;
			packedKernelArgs[deviceid][4]=(void*)&nptr;
			packedKernelArgs[deviceid][5]=(void*)&tile;
			
			launchParamsList[deviceid].func=(void*)kernel_func;
			launchParamsList[deviceid].gridDim=blockPerGPU;
			launchParamsList[deviceid].blockDim=threadPerBlock;
			launchParamsList[deviceid].sharedMem=32;
			launchParamsList[deviceid].stream=mstream[deviceid];
			launchParamsList[deviceid].args=packedKernelArgs[deviceid];
		}
		cudaCheckError(); 

		timespec tsstart,tsendop;
		long time_elapsed_ns ;
		double latency_lat[SIZE];
		
		for(int i=0; i<SIZE; i++)
		{

			clock_gettime(CLOCK_REALTIME, &tsstart);
			run_func(kernel_func,blockPerGPU,threadPerBlock,NULL, gpu_count,launchParamsList);
			for(int deviceid=0; deviceid<gpu_count; deviceid++)
			{
				hipSetDevice(deviceid);
				hipDeviceSynchronize();
				hipStreamSynchronize(mstream[deviceid]);
			}
 			clock_gettime(CLOCK_REALTIME, &tsendop);

	 		//latencys of total kernel total latency (after sync)
			time_elapsed_ns = (tsendop.tv_nsec-tsstart.tv_nsec);
	 		time_elapsed_ns += 1000000000*(tsendop.tv_sec-tsstart.tv_sec);
	 		latency_lat[i]=time_elapsed_ns;
	 		e=hipGetLastError();                                 
 			if(e!=hipSuccess) {                                              
	   			fprintf(stderr,"Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); 
	   			for(int deviceid=0; deviceid<gpu_count;deviceid++)
				{
					hipSetDevice(deviceid);	
		 			hipDeviceReset();
				}
				errorcode=-1;
				break;
	 		}

		}
		// cudaCheckError();
		getStatistics(result->mean_lat, result->s_lat, latency_lat+1, SIZE-1);

		for(int deviceid=0; deviceid<gpu_count;deviceid++)
		{
			hipSetDevice(deviceid);	
 			hipStreamDestroy(mstream[deviceid]);
			hipFree(d_out[deviceid]);
		}

		free(mstream);
		free(packedKernelArgs);
		free(launchParamsList);
		free(d_out);
		// e=hipGetLastError();                                 
 	// 	if(e!=hipSuccess) {                                              
  //  			fprintf(stderr,"Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); 
		// 	errorcode=-1;
 	// 	}
 		if(errorcode!=1)
 		{
 			for(int deviceid=0; deviceid<gpu_count;deviceid++)
			{
				hipSetDevice(deviceid);	
	 			hipDeviceReset();
			}
			return -1;
 		}
 		return 1;
	}
}
//3. mearsure kernel latency 

template <int gpu_count>
void measureKernelLatency(latencys* result, 
	launchfunction_nkernel run_func, nKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock)
{
	{
		hipStream_t *mstream = (hipStream_t*)malloc(sizeof(hipStream_t)*gpu_count);
		void***packedKernelArgs = (void***)malloc(sizeof(void**)*gpu_count); 
		hipLaunchParams *launchParamsList = (hipLaunchParams *)malloc(
      		sizeof(hipLaunchParams)*gpu_count);

		for(int deviceid=0; deviceid<gpu_count;deviceid++)
		{
			hipSetDevice(deviceid);
			packedKernelArgs[deviceid]=(void**)malloc(sizeof(void*));

			hipStreamCreate(&mstream[deviceid]);

			cudaCheckError();

			packedKernelArgs[deviceid][0]=NULL;
			
			launchParamsList[deviceid].func=(void*)kernel_func;
			launchParamsList[deviceid].gridDim=blockPerGPU;
			launchParamsList[deviceid].blockDim=threadPerBlock;
			launchParamsList[deviceid].sharedMem=32;
			launchParamsList[deviceid].stream=mstream[deviceid];
			launchParamsList[deviceid].args=packedKernelArgs[deviceid];
		}
		cudaCheckError(); 

		timespec ini,tsstart,tsend,tsendop,tsendsync;
		long time_elapsed_ns;
		double latency_laun[SIZE];
		double latency_lat[SIZE];
		double latency_clock[SIZE];
		double latency_syncfunc[SIZE];
		
		for(int i=0; i<SIZE; i++)
		{
			//clock
			clock_gettime(CLOCK_REALTIME, &ini);
			clock_gettime(CLOCK_REALTIME, &tsstart);
			//launch
			run_func(kernel_func,blockPerGPU,threadPerBlock,gpu_count,launchParamsList);
			clock_gettime(CLOCK_REALTIME, &tsend);
			//execution
			if(gpu_count==0)
			{
				hipDeviceSynchronize();
			}
			else
			{
				for(int deviceid=0; deviceid<gpu_count; deviceid++)
				{
					hipSetDevice(deviceid);
					hipDeviceSynchronize();
					hipStreamSynchronize(mstream[deviceid]);
				}
			}
 			clock_gettime(CLOCK_REALTIME, &tsendop);
			if(gpu_count==0)
			{
				hipDeviceSynchronize();
			}
			else
			{
				for(int deviceid=0; deviceid<gpu_count; deviceid++)
				{
					hipSetDevice(deviceid);
					hipDeviceSynchronize();
					hipStreamSynchronize(mstream[deviceid]);
				}
			}
 			clock_gettime(CLOCK_REALTIME, &tsendsync);
 			//latencys of clock function
	 		time_elapsed_ns = (tsstart.tv_nsec-ini.tv_nsec);
	 		time_elapsed_ns += 1000000000*(tsstart.tv_sec-ini.tv_sec);
	 		latency_clock[i]=time_elapsed_ns;

	 		//latencys of launch functions (no sync here)
	 		time_elapsed_ns = (tsend.tv_nsec-tsstart.tv_nsec);
	 		time_elapsed_ns += 1000000000*(tsend.tv_sec-tsstart.tv_sec);
	 		latency_laun[i]=time_elapsed_ns;

	 		//latencys of total kernel total latency (after sync)
			time_elapsed_ns = (tsendop.tv_nsec-tsstart.tv_nsec);
	 		time_elapsed_ns += 1000000000*(tsendop.tv_sec-tsstart.tv_sec);
	 		latency_lat[i]=time_elapsed_ns;

	 		//latencys of synchronization functions 
			time_elapsed_ns = (tsendsync.tv_nsec-tsendop.tv_nsec);
	 		time_elapsed_ns += 1000000000*(tsendsync.tv_sec-tsendop.tv_sec);
	 		latency_syncfunc[i]=time_elapsed_ns;
		}
		cudaCheckError();

		getStatistics(result->mean_laun, result->s_laun, latency_laun+1, SIZE-1);
		getStatistics(result->mean_clk, result->s_clk, latency_clock+1, SIZE-1);
		getStatistics(result->mean_lat, result->s_lat, latency_lat+1, SIZE-1);
		getStatistics(result->mean_sync, result->s_sync, latency_syncfunc+1, SIZE-1);

		for(int deviceid=0; deviceid<gpu_count;deviceid++)
		{
			hipSetDevice(deviceid);	
			cudaCheckError();
			hipStreamDestroy(mstream[deviceid]);
		}

		free(mstream);
		free(packedKernelArgs);
		free(launchParamsList);
	}
}

template void measureKernelLatency<1>(latencys* result, launchfunction_nkernel run_func, nKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock);
template void measureKernelLatency<2>(latencys* result, launchfunction_nkernel run_func, nKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock);
template void measureKernelLatency<3>(latencys* result, launchfunction_nkernel run_func, nKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock);
template void measureKernelLatency<4>(latencys* result, launchfunction_nkernel run_func, nKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock);
template void measureKernelLatency<5>(latencys* result, launchfunction_nkernel run_func, nKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock);
template void measureKernelLatency<6>(latencys* result, launchfunction_nkernel run_func, nKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock);
template void measureKernelLatency<7>(latencys* result, launchfunction_nkernel run_func, nKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock);
template void measureKernelLatency<8>(latencys* result, launchfunction_nkernel run_func, nKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock);
