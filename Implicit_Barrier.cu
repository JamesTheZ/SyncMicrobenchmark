#include "repeat.h"
// #include "Implicit_Barrier_Kernel.cuh"
#include "Implicit_Barrier.h"
#include "util.h"

#include <stdio.h>
/*
Definition:
* {Kernel Execution Latency:} Total time spent in executing the kernel, excluding any overhead for launching the kernel.
* {Launch Overhead:} Latency that is not related to kernel execution. 
* {Kernel Total Latency:} Total latency to run kernels.
Depends on different situation, the launch overhead could be different:
Situation 1: Launch a single kernel
Situation 2: Launch additional "small kenel" (By "small" we mean the device is not saturate at all, in this experiment in single GPU if each kernel lasts less then 5us it is defined as "small")
Situation 3: Launch additional "big kernel" (By "big" we mean the device is saturate enough while the workload is not severe, in this experiment in single GPU if each kernel lasts longer than 5us, it is defined as "big")

When kernels are "small" or each kernel lasts fewer than 5us, it would not be practical to offload these workloads to GPU at all. So, we only include the launch overhead of "big kernel" in our IPDPS20 paper.

But in this microbenchmark, we include the measurements of launch overhead in all three situations. The detailed information about these measurements are explained in an ICPP19 Poster in this same folder. 

*/

void workload_influence()
{
	//increase the workload only test the additional latency.

}


int main(int argc, char **argv)
{
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    cudaCheckError();
   	unsigned int smx_count = deviceProp.multiProcessorCount;
//	double* result=(double*)malloc(sizeof(double)*6*8);
	//show how total latency is influenced by execution (traditional launch)


	//merge this two situation together
	//launch single null kernel and different features
	//launch additional null kernel and compute the kernel overhead here
	
	// Test_Null_Kernel(smx_count,1024);
	// Test_Null_Kernel_MGPU<2>(1,32);
	
	//launch big kernel and additional big kernel to compute the kernel overhead
	Test_Sleep_Kernel(smx_count,1024);
	Test_Sleep_Kernel_MGPU<8>(smx_count,1024);

	Test_Workload_Influence(smx_count,1024);
}



