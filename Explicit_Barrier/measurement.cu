#include "hip/hip_runtime.h"

#include"measurement.cuh"
#include "wrap_launch_functions.cuh"
#include "../share/util.h"

#include <stdio.h>


//1. measure latencys in cycle (single GPU)
//1.1. measure the latency of instructions in the first SM
//1.2. measure the throughput of instructions in the first SM
void measureLatency_cycle(latencys* result, 
	launchfunction run_func, fbaseKernel kernel_func,
	unsigned int blockPerGPU, unsigned int threadPerBlock, 
	float a, float b, unsigned int tile)
{
	{
		double * d_out;
		unsigned int totalThreadsPerGPU = blockPerGPU*threadPerBlock;
		hipMalloc((void **)&d_out, sizeof(double) * totalThreadsPerGPU*1);	
		unsigned int warp_count=blockPerGPU*threadPerBlock/32;

	 	unsigned int * h_time_stamp = (unsigned int*)malloc(sizeof(unsigned int)*warp_count*2); 
	 	unsigned int * d_time_stamp;
	 	hipMalloc((void**)&d_time_stamp, sizeof(unsigned int)*warp_count*2);

	 	unsigned int * h_idx = (unsigned int*)malloc(sizeof(unsigned int)*warp_count*2); 
	 	unsigned int * d_idx;
	 	hipMalloc((void**)&d_idx, sizeof(unsigned int)*warp_count*2);

		void*KernelArgs[] = {(void*)&a, 
						(void*)&b,
						(void*)&d_out,
						(void*)&d_time_stamp,
						(void*)&d_idx,
						(void*)&tile};

		cudaCheckError(); 

		timespec tsstart,tsend;
		long time_elapsed_ns_lat ;
		double latency_lat[SIZE];
		double latency_max[SIZE];
		double latency_min[SIZE];
		unsigned int ulatency_max;
		unsigned int ulatency_min;
		for(int i=0; i<SIZE; i++)
		{
			//clock
			clock_gettime(CLOCK_REALTIME, &tsstart);
			//launch
			run_func(kernel_func,blockPerGPU,threadPerBlock,KernelArgs,1,NULL);
			hipDeviceSynchronize();
			clock_gettime(CLOCK_REALTIME, &tsend);
			//execution

			hipMemcpy(h_time_stamp, d_time_stamp, sizeof(unsigned int)*warp_count*2, hipMemcpyDeviceToHost);
			hipMemcpy(h_idx, d_idx, sizeof(unsigned int)*warp_count*2, hipMemcpyDeviceToHost);

	 		time_elapsed_ns_lat = (tsend.tv_nsec-tsstart.tv_nsec);
	 		time_elapsed_ns_lat += 1000000000*(tsend.tv_sec-tsstart.tv_sec);
	 		latency_lat[i]=time_elapsed_ns_lat;

	 		GetLatencyOfSM(ulatency_min,ulatency_max,warp_count,h_time_stamp,h_idx,0);
	 		latency_min[i]=ulatency_min;
	 		latency_max[i]=ulatency_max;
		}
		cudaCheckError();

		getStatistics(result->mean_lat, result->s_lat, latency_lat+1, SIZE-1);
		getStatistics(result->latency_min, result->s_latency_min, latency_min+1, SIZE-1);
		getStatistics(result->latency_max, result->s_latency_max, latency_max+1, SIZE-1);

		hipFree(d_out);
		hipFree(d_time_stamp);
		hipFree(d_idx);
		free(h_time_stamp);
		free(h_idx);
	}
}
//2. measure latencys in ns (involve several SMs) TODO


// typedef void(*launchfunction)(nKernel, unsigned int, unsigned int, unsigned int, hipLaunchParams*);
// template <int gpu_count>
// void measureLatencys(latencys* result, launchfunction run_func, 
// 	fbaseKernel kernel_func,
// 	unsigned int blockPerGPU, unsigned int threadPerBlock)
// {
// 	{
// 		hipStream_t *mstream = (hipStream_t*)malloc(sizeof(hipStream_t)*gpu_count);
// 		void***packedKernelArgs = (void***)malloc(sizeof(void**)*gpu_count); 
// 		hipLaunchParams *launchParamsList = (hipLaunchParams *)malloc(
//       		sizeof(hipLaunchParams)*gpu_count);

// 		for(int deviceid=0; deviceid<gpu_count;deviceid++)
// 		{
// 			hipSetDevice(deviceid);
// 			packedKernelArgs[deviceid]=(void**)malloc(sizeof(void*));

// 			hipStreamCreate(&mstream[deviceid]);

// 			cudaCheckError();

// 			packedKernelArgs[deviceid][0]=NULL;
			
// 			launchParamsList[deviceid].func=(void*)kernel_func;
// 			launchParamsList[deviceid].gridDim=blockPerGPU;
// 			launchParamsList[deviceid].blockDim=threadPerBlock;
// 			launchParamsList[deviceid].sharedMem=32;
// 			launchParamsList[deviceid].stream=mstream[deviceid];
// 			launchParamsList[deviceid].args=packedKernelArgs[deviceid];
// 		}
// 		cudaCheckError(); 

// 		timespec ini,tsstart,tsend,tsendop,tsendsync;
// 		long time_elapsed_ns_laun, time_elapsed_ns_lat ;
// 		double latency_laun[SIZE];
// 		double latency_lat[SIZE];
// 		double latency_clock[SIZE];
// 		double latency_syncfunc[SIZE];
		
// 		for(int i=0; i<SIZE; i++)
// 		{
// 			//clock
// 			clock_gettime(CLOCK_REALTIME, &ini);
// 			clock_gettime(CLOCK_REALTIME, &tsstart);
// 			//launch
// 			run_func(kernel_func,blockPerGPU,threadPerBlock,gpu_count,launchParamsList);
// 			clock_gettime(CLOCK_REALTIME, &tsend);
// 			//execution
// 			if(gpu_count==0)
// 			{
// 				hipDeviceSynchronize();
// 			}
// 			else
// 			{
// 				for(int deviceid=0; deviceid<gpu_count; deviceid++)
// 				{
// 					hipSetDevice(deviceid);
// 					hipDeviceSynchronize();
// 					hipStreamSynchronize(mstream[deviceid]);
// 				}
// 			}
//  			clock_gettime(CLOCK_REALTIME, &tsendop);
// 			if(gpu_count==0)
// 			{
// 				hipDeviceSynchronize();
// 			}
// 			else
// 			{
// 				for(int deviceid=0; deviceid<gpu_count; deviceid++)
// 				{
// 					hipSetDevice(deviceid);
// 					hipDeviceSynchronize();
// 					hipStreamSynchronize(mstream[deviceid]);
// 				}
// 			}
//  			clock_gettime(CLOCK_REALTIME, &tsendsync);
//  			//latencys of clock function
// 	 		time_elapsed_ns_laun = (tsstart.tv_nsec-ini.tv_nsec);
// 	 		time_elapsed_ns_laun += 1000000000*(tsstart.tv_sec-ini.tv_sec);
// 	 		latency_clock[i]=time_elapsed_ns_laun;

// 	 		//latencys of launch functions (no sync here)
// 	 		time_elapsed_ns_laun = (tsend.tv_nsec-tsstart.tv_nsec);
// 	 		time_elapsed_ns_laun += 1000000000*(tsend.tv_sec-tsstart.tv_sec);
// 	 		latency_laun[i]=time_elapsed_ns_laun;

// 	 		//latencys of total kernel total latency (after sync)
// 			time_elapsed_ns_lat = (tsendop.tv_nsec-tsstart.tv_nsec);
// 	 		time_elapsed_ns_lat += 1000000000*(tsendop.tv_sec-tsstart.tv_sec);
// 	 		latency_lat[i]=time_elapsed_ns_lat;

// 	 		//latencys of synchronization functions 
// 			time_elapsed_ns_lat = (tsendsync.tv_nsec-tsendop.tv_nsec);
// 	 		time_elapsed_ns_lat += 1000000000*(tsendsync.tv_sec-tsendop.tv_sec);
// 	 		latency_syncfunc[i]=time_elapsed_ns_lat;
// 		}
// 		cudaCheckError();

// 		getStatistics(result->mean_laun, result->s_laun, latency_laun+1, SIZE-1);
// 		getStatistics(result->mean_clk, result->s_clk, latency_clock+1, SIZE-1);
// 		getStatistics(result->mean_lat, result->s_lat, latency_lat+1, SIZE-1);
// 		getStatistics(result->mean_sync, result->s_sync, latency_syncfunc+1, SIZE-1);

// 		for(int deviceid=0; deviceid<gpu_count;deviceid++)
// 		{
// 			hipSetDevice(deviceid);	
// 			cudaCheckError();
// 			hipStreamDestroy(mstream[deviceid]);
// 		}

// 		free(mstream);
// 		free(packedKernelArgs);
// 		free(launchParamsList);
// 	}
// }

// template void measureLatencys<1>(latencys* result, launchfunction run_func, nKernel kernel_func,
// 	unsigned int blockPerGPU, unsigned int threadPerBlock);
// template void measureLatencys<2>(latencys* result, launchfunction run_func, nKernel kernel_func,
// 	unsigned int blockPerGPU, unsigned int threadPerBlock);
// template void measureLatencys<3>(latencys* result, launchfunction run_func, nKernel kernel_func,
// 	unsigned int blockPerGPU, unsigned int threadPerBlock);
// template void measureLatencys<4>(latencys* result, launchfunction run_func, nKernel kernel_func,
// 	unsigned int blockPerGPU, unsigned int threadPerBlock);
// template void measureLatencys<5>(latencys* result, launchfunction run_func, nKernel kernel_func,
// 	unsigned int blockPerGPU, unsigned int threadPerBlock);
// template void measureLatencys<6>(latencys* result, launchfunction run_func, nKernel kernel_func,
// 	unsigned int blockPerGPU, unsigned int threadPerBlock);
// template void measureLatencys<7>(latencys* result, launchfunction run_func, nKernel kernel_func,
// 	unsigned int blockPerGPU, unsigned int threadPerBlock);
// template void measureLatencys<8>(latencys* result, launchfunction run_func, nKernel kernel_func,
// 	unsigned int blockPerGPU, unsigned int threadPerBlock);