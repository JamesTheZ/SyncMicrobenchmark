#include "hip/hip_runtime.h"
#include "Implicit_Barrier_Kernel.cuh"
#include "Implicit_Barrier.h"
#include "wrap_launch_functions.cuh"
#include "util.h"
#include "measurement.cuh"

#include <stdio.h>


__global__ void null_kernel(){}


//In order to reduce overhead, we use repeat MACRO instead of forloop here. 
//The inconvenience part is that when we need to test overhead, we would need to introduce additional MACRO
#define NULL_KERNEL_TEST(callfunc, basicDEP, moreDEP, gpu_count) \
	printf("method\tGPUcount\trep\tblk\tthrd\tm(clk)\ts(clk)\tm(sync)\ts(sync)\tm(laun)\ts(laun)\tm(ttl)\ts(ttl)\tm(avelaun)\ts(avelaun)\tm(addl)\ts(addl)\n");\
	measureLatencys<gpu_count>(result, callfunc##_##basicDEP, null_kernel,block_perGPU,thread_perBlock);\
	printf("%s\t%s\t%s\t%u\t%u\t",#callfunc,#gpu_count,#basicDEP,block_perGPU,thread_perBlock);\
	showlatency(result);printf("%f\t%f\t",result[0].mean_laun/basicDEP,result[0].s_laun/basicDEP);nxtline();\
	measureLatencys<gpu_count>(result+1, callfunc##_##moreDEP, null_kernel,block_perGPU,thread_perBlock);\
	printf("%s\t%s\t%s\t%u\t%u\t",#callfunc,#gpu_count,#moreDEP,block_perGPU,thread_perBlock);\
	showlatency(result+1);printf("%f\t%f\t",result[1].mean_laun/moreDEP,result[1].s_laun/moreDEP);printf("%f\t%f\t",computeAddLat(result,moreDEP-basicDEP),computeAddLats(result,moreDEP-basicDEP));nxtline();


void Null_Kernel(unsigned int block_perGPU, unsigned int thread_perBlock)
{
	latencys* result  = (latencys*)malloc(2*sizeof(latencys));

	printf("Empty Kernel\n");
	printf("When Calling count is one, the result of total latency (ns)\n");

	NULL_KERNEL_TEST(traditional_launch,1,128,1);
	NULL_KERNEL_TEST(cooperative_launch,1,128,1);

	free(result);
}


#define NULL_KERNEL_TEST_8GPU(callfunc, basicDEP,moreDEP) \
	if(gpu_count>=1)\
	{\
		NULL_KERNEL_TEST(callfunc, 1, 128, 1);\
	}\
	if(gpu_count>=2)\
	{\
		NULL_KERNEL_TEST(callfunc, 1, 128, 2);\
	}\
	if(gpu_count>=3)\
	{\
		NULL_KERNEL_TEST(callfunc, 1, 128, 3);\
	}\
	if(gpu_count>=4)\
	{\
		NULL_KERNEL_TEST(callfunc, 1, 128, 4);\
	}\
	if(gpu_count>=5)\
	{\
		NULL_KERNEL_TEST(callfunc, 1, 128, 5);\
	}\
	if(gpu_count>=6)\
	{\
		NULL_KERNEL_TEST(callfunc, 1, 128, 6);\
	}\
	if(gpu_count>=7)\
	{\
		NULL_KERNEL_TEST(callfunc, 1, 128, 7);\
	}\
	if(gpu_count>=8)\
	{\
		NULL_KERNEL_TEST(callfunc, 1, 128, 8);\
	}\

template <int gpu_count>
void Null_Kernel_MGPU(unsigned int block_perGPU, unsigned int thread_perBlock)
{

	printf("Empty Kernel for multi-GPU\n");
	printf("When Calling count is one, the result of total latency (ns)\n");

	latencys* result  = (latencys*)malloc(2*sizeof(latencys));
	
	NULL_KERNEL_TEST_8GPU(multi_cooperative_launch,1,128);
	NULL_KERNEL_TEST_8GPU(omp_traditional_launch,1,128);

	free(result);
}


template void Null_Kernel_MGPU<1>(unsigned int block_perGPU, unsigned int thread_perBlock);
template void Null_Kernel_MGPU<2>(unsigned int block_perGPU, unsigned int thread_perBlock);
template void Null_Kernel_MGPU<3>(unsigned int block_perGPU, unsigned int thread_perBlock);
template void Null_Kernel_MGPU<4>(unsigned int block_perGPU, unsigned int thread_perBlock);
template void Null_Kernel_MGPU<5>(unsigned int block_perGPU, unsigned int thread_perBlock);
template void Null_Kernel_MGPU<6>(unsigned int block_perGPU, unsigned int thread_perBlock);
template void Null_Kernel_MGPU<7>(unsigned int block_perGPU, unsigned int thread_perBlock);
template void Null_Kernel_MGPU<8>(unsigned int block_perGPU, unsigned int thread_perBlock);


// int main(int argc, char **argv)
// {
// 	hipDeviceProp_t deviceProp;
//     hipGetDeviceProperties(&deviceProp, 0);
//     cudaCheckError();
//    	unsigned int smx_count = deviceProp.multiProcessorCount;

// 	Null_Kernel(smx_count,1024);
// 	Null_Kernel_MGPU<2>(1,32);

// }
